#include "hip/hip_runtime.h"
#include "Structures.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/geometry/SparseGrid.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/zpc_tpls/fmt/color.h"
#include "zensim/zpc_tpls/fmt/format.h"

#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>

// #include <zeno/VDBGrid.h>

#include "../utils.cuh"
#include "zeno/utils/log.h"

namespace zeno {

struct ZSSParseGridDifference : INode {
  void apply() override {
    using namespace zs;
    auto grid = get_input<ZenoSparseGrid>("ZSGrid");

    auto attrTag = get_input2<std::string>("attrName");
    auto chnOffset = get_input2<int>("channelOffset");

    auto outputAttrTag = get_input2<std::string>("outputAttrName");
    if (outputAttrTag.empty())
      throw std::runtime_error(
          "[outputAttrName] should not be an empty string.");

    auto orientationStr = get_input2<std::string>("orientation");
    int orientation =
        orientationStr == "ddx" ? 0 : (orientationStr == "ddy" ? 1 : 2);

    auto boundaryStr = get_input2<std::string>("boundary_type");
    int boundaryType = boundaryStr == "neumann" ? 0 : /*dirichlet*/ 1;

    auto &spg = grid->spg;
    auto block_cnt = spg.numBlocks();
    auto pol = cuda_exec();
    constexpr auto space = execspace_e::cuda;

    spg.append_channels(pol, {{outputAttrTag, 1}});

    pol(Collapse{block_cnt, spg.block_size},
        [spgv = proxy<space>(spg),
         srcOffset = spg.getPropertyOffset(attrTag) + chnOffset, orientation,
         dstOffset = spg.getPropertyOffset(outputAttrTag), boundaryType,
         twodx = 2 * spg.voxelSize()[0]] __device__(int blockno,
                                                    int cellno) mutable {
          auto icoord = spgv.iCoord(blockno, cellno);
          auto val = spgv(srcOffset, blockno, cellno);
          auto iCoordA = icoord;
          iCoordA[orientation]++;
          auto iCoordB = icoord;
          iCoordB[orientation]--;

          auto getVal = [&](const auto &coord) -> zs::f32 {
            auto [bno, cno] = spgv.decomposeCoord(coord);
            if (bno == spgv.sentinel_v) {
              // boundary
              if (boundaryType == 0) // neumann
                return val;
              else
                return 0.f;
            } else {
              return spgv(srcOffset, bno, cno);
            }
          };
          auto tmp = (getVal(iCoordA) - getVal(iCoordB)) / twodx;
#if 0
          if (zs::abs(tmp) > 0.01) {
            printf("coord (%d, %d, %d) - (%d, %d, %d) diff: %f\n", iCoordA[0],
                   iCoordA[1], iCoordA[2], iCoordB[0], iCoordB[1], iCoordB[2],
                   (float)tmp);
          }
#endif
          spgv(dstOffset, blockno, cellno) = tmp;
        });

    set_output("ZSGrid", grid);
  }
};

ZENDEFNODE(ZSSParseGridDifference,
           {/* inputs: */
            {
                "ZSGrid",
                {"string", "attrName", "sdf"},
                {"int", "channelOffset", "0"},
                {"enum ddx ddy ddz", "orientation", "ddx"},
                {"string", "outputAttrName", ""},
                {"enum neumann dirichlet", "boundary_type", "neumann"},
            },
            /* outputs: */
            {"ZSGrid"},
            /* params: */
            {},
            /* category: */
            {"Eulerian"}});

} // namespace zeno